#include "hip/hip_runtime.h"
#include <cub/warp/warp_reduce.cuh>
#include <cudf/column/column.hpp>

using namespace cudf;

/*
template<size_type block_size>
void compute_validity(size_type* valid_count, size_type const tid, bool const valid)
{
  // compute null count for the block. each warp processes one string, so lane 0
  // from each warp contributes 1 bit of validity
  size_type const block_valid_count = cudf::detail::single_lane_block_sum_reduce<block_size, 0>(valid ? 1 : 0);
  if (tid == 0) { atomicAdd(valid_count, block_valid_count); }
}
*/

template<typename T, size_type block_size>
__global__ void string_to_float_kernel(T* out,
                                       bitmask_type* validity,
                                       int32_t *ansi_except,
                                       size_type* valid_count,
                                       const char* const chars,
                                       offset_type const* offsets,
                                       size_type num_rows)
{
  size_type const tid = threadIdx.x + (blockDim.x * blockIdx.x);
  size_type const warp_id = tid / 32;
  size_type const row = warp_id;
  if(row >= num_rows){
    return;
  }
  size_type const warp_lane = tid % 32; 
  size_type const row_start = offsets[row];
  size_type const len = offsets[row+1] - row_start;   
  
  size_type bstart = 0;                             // start position of the current batch
  size_type blen = min(32, len);                    // length of the batch
  size_type bpos = 0;                               // current position within the current batch of chars for the warp  
  char c = warp_lane < blen ? chars[row_start + warp_lane] : 0;

  size_type tpos = warp_lane;                       // current thread position relative to bpos  

  // printf("(%d): bstart(%d), blen(%d), bpos(%d), tpos(%d), c(%c)\n", tid, bstart, blen, bpos, tpos, c);

  __shared__ uint64_t ipow[19];
  if(threadIdx.x == 0){
    ipow[0] = 1;
    ipow[1] = 10;
    ipow[2] = 100;
    ipow[3] = 1000;
    ipow[4] = 10000;
    ipow[5] = 100000;
    ipow[6] = 1000000;
    ipow[7] = 10000000;
    ipow[8] = 100000000;
    ipow[9] = 1000000000;
    ipow[10] = 10000000000;
    ipow[11] = 100000000000;
    ipow[12] = 1000000000000;
    ipow[13] = 10000000000000;
    ipow[14] = 100000000000000;
    ipow[15] = 1000000000000000;
    ipow[16] = 10000000000000000;
    ipow[17] = 100000000000000000;
    ipow[18] = 1000000000000000000; 
  }
  __syncthreads();

  // a valid string can -only- start with:
  // nan
  // OR
  // +/-
  // inf
  // infinity
  // digits

  // check for leading nan
  auto const nan_mask = __ballot_sync(0xffffffff, (tpos == 0 && (c == 'N' || c == 'n')) ||
                                                  (tpos == 1 && (c == 'A' || c == 'a')) ||
                                                  (tpos == 2 && (c == 'N' || c == 'n')));
  if(nan_mask == 0x7){    
    // if we start with 'nan', then even if we have other garbage character, this is a null row.
    //
    // if we're in ansi mode and this is not -precisely- nan, report that so that we can throw
    // an exception later.
    if(warp_lane == 0 && ansi_except && len != 3){
      atomicOr(ansi_except, 1);
    }
    //compute_validity(tid, false);    
    return;
  }
    
  // check for + or -
  auto const sign_mask = __ballot_sync(0xffffffff, tpos == 0 && (c == '+' || c == '-'));
  int sign = 1;
  if(sign_mask){
    bpos++;
    tpos--;
    sign = c == '+' ? 1 : -1;
  }

  // check for inf or infinity
  auto const inf_mask = __ballot_sync(0xffffffff, (tpos == 0 && (c == 'I' || c == 'i')) ||
                                                  (tpos == 1 && (c == 'N' || c == 'n')) ||
                                                  (tpos == 2 && (c == 'F' || c == 'f')) );
  if(inf_mask == 0x7){
    bpos += 3;
    tpos -= 3;
    
    // if we're at the end
    if(bpos == len){
      if(warp_lane == 0){
        out[row] = sign > 0? std::numeric_limits<T>::infinity() : -std::numeric_limits<T>::infinity();
      }
      //compute_validity(tid, true);    
      return;
    }

    // see if we have the whole word
    auto const infinity_mask = __ballot_sync(0xffffffff, (tpos == 0 && (c == 'I' || c == 'i')) ||
                                                         (tpos == 1 && (c == 'N' || c == 'n')) ||
                                                         (tpos == 2 && (c == 'I' || c == 'i')) ||
                                                         (tpos == 3 && (c == 'T' || c == 't')) ||
                                                         (tpos == 4 && (c == 'Y' || c == 'y')));
    if(infinity_mask == 0x1f){
      // if we're at the end
      if(bpos == len){
        if(warp_lane == 0){
          out[row] = sign > 0 ? std::numeric_limits<T>::infinity() : -std::numeric_limits<T>::infinity();
        }
        //compute_validity(tid, true);
        return;
      }
    }

    // if we reach here for any reason, it means we have "inf" or "infinity" at the start of the string but
    // also have additional characters, making this whole thing bogus/null
    if(warp_lane == 0 && ansi_except){
      atomicOr(ansi_except, 1);
    }
    //compute_validity(tid, false);
    return;
  }

  // parse the remainder as (potentially) valid floating point. 

  // shuffle remaining chars down so lane 0 has the first unprocessed digit of the batch
  c = __shfl_down_sync(0xffffffff, c, bpos);
  
  typedef hipcub::WarpReduce<uint64_t> WarpReduce;
  __shared__ typename WarpReduce::TempStorage temp_storage;
      
  int total_digits = 0;
  uint64_t digits = 0;
  int exp_ten = 0;
  int decimal_pos = 0;
  bool decimal = false;
  int count = 0;
  constexpr int max_safe_digits = 19;
  bool truncating = false;  
  do {    
    int num_chars = min(max_safe_digits, blen - (bstart + bpos));
    if(warp_lane == 0){
      printf("NC: %d (%d, %d, %d)\n", num_chars, blen, bstart, bpos);
    }

    // if our current sum is 0 and we don't have a decimal, strip leading
    // zeros.  handling cases such as
    // 0000001
    if(!decimal && digits == 0){
      auto const zero_mask = __ballot_sync(0xffffffff, warp_lane < num_chars && c != '0');
      auto const nz_pos = __ffs(zero_mask) - 1;
      if(nz_pos > 0){
        num_chars -= nz_pos;
        bpos += nz_pos;
        c = __shfl_down_sync(0xffffffff, c, nz_pos);
      }
    }

    // # of digits to the left and right of the decimal for this iteration
    int ldigits = decimal ? 0 : num_chars;
    int rdigits = num_chars - ldigits;

    // handle a decimal point    
    auto const decimal_mask = __ballot_sync(0xffffffff, warp_lane < num_chars && c == '.');    
    if(decimal_mask){
      // if we have more than one decimal, this is an invalid value
      if(decimal || __popc(decimal_mask) > 1){
        //
      }   
      auto const dpos = __ffs(decimal_mask)-1;    // 0th bit is reported as 1 by __ffs
      decimal_pos = (dpos + total_digits);            
      decimal = true;      

      // strip the decimal char out
      if(warp_lane >= dpos){
        c = __shfl_down_sync(~((1 << dpos) - 1), c, 1);
      }
      num_chars--;
      ldigits = dpos;
      rdigits = num_chars - ldigits;
    }    

    // our local digit
    uint64_t const digit = warp_lane < num_chars ? static_cast<uint64_t>(c - '0') * ipow[(num_chars - warp_lane) - 1] : 0;
          
    // we may have to start truncating because we'd go past the 64 bit limit by adding the new digits
    //
    // max uint64_t is 20 digits, so any 19 digit number is valid.
    // 2^64:  18,446,744,073,709,551,616
    //         9,999,999,999,999,999,999
    //
    // if the 20th digit would push us past that limit, we have to start truncating.
    // max_holding:  1,844,674,407,370,955,160
    // so     1,844,674,407,370,955,160 ++ 9    -> 18,446,744,073,709,551,609  -> ok
    //        1,844,674,407,370,955,160 ++ 1X   -> 18,446,744,073,709,551,61X  -> potentially rolls past the limit
    //
    constexpr uint64_t max_holding = (std::numeric_limits<uint64_t>::max() - 9) / 10;
    // if we're already past the max_holding, just truncate.
    // eg:    9,999,999,999,999,999,999
    if(digits > max_holding){
      if(warp_lane == 0){
        printf("A\n");
      }      
      exp_ten += ldigits;
    } 
    else {
      // add as many digits to the running sum as we can.
      int const safe_count = min(max_safe_digits - total_digits, num_chars);
      if(safe_count > 0){
        // only lane 0 will have the real value so we need to shfl it to the rest of the threads.
        digits = (digits * ipow[safe_count]) + __shfl_sync(0xffffffff, WarpReduce(temp_storage).Sum(digit, safe_count), 0);
        total_digits += safe_count;

        if(warp_lane == 0){
          printf("B: total_digits(%d)\n", total_digits);
        }
      }

      // if we have more digits
      if(safe_count < num_chars){
        // we're already past max_holding so we have to start truncating
        if(digits > max_holding){
          if(warp_lane == 0){
            printf("C\n");
          }
        } 
        // we may be able to add one more digit.
        else {
          auto const last_digit = static_cast<uint64_t>(__shfl_sync(0xffffffff, c, safe_count) - '0');
          if((digits * 10) + last_digit <= max_holding){
            // we can add this final digit
            digits = (digits * 10) + last_digit;

            if(warp_lane == 0){
              printf("D\n");
            }
          }
          // everything else gets truncated

          if(warp_lane == 0){
            printf("E\n");
          }
        }
      }
    } 
    bpos += num_chars + (decimal_mask > 0);        
    // adjust the exponent        
    if(decimal){
      printf("EA\n");
      // move left for every digit to the right of the decimal
      exp_ten += ((decimal ? -1 : 0) * num_chars) + (decimal_mask > 0 ? decimal_pos : 0);
    } 
    // if we are to the left of the decimal, we're just truncating extra digits and increasing
    // the exponent instead.
    else if(digits > max_holding){    
      printf("EB\n");
      exp_ten += num_chars;
    }    
    if(warp_lane == 0){
      printf("EXPT: %d (%d, %d, %d, %d)\n", exp_ten, decimal ? 1 : 0, num_chars, decimal_mask, decimal_pos);
    }
    
    /*
    if(warp_lane == 0){   
      printf("EO: %d -> %d (%d, %d, %d)\n", exp_ten - (decimal * num_chars), exp_ten, decimal, num_chars);
    }
    */

    // read the next batch of chars.
    if(bpos == blen){
      bstart += blen;
      // nothing left to read?
      if(bstart == len){
        break;
      }
      // read the next batch
      bpos = 0;
      blen = min(32, len - bstart);
      char c = warp_lane < blen ? chars[row_start + bstart + warp_lane] : 0;
    } else {
      c = __shfl_down_sync(0xffffffff, c, bpos);
    }
  } while(1);
  
  if(warp_lane == 0){
    // 0 / -0
    if(digits == 0){
      out[row] = sign * static_cast<double>(0);
      return;
    }

    // base value
    double digitsf = sign * static_cast<double>(digits);
    
    // exponent    
    printf("ET: %d, %d\n", exp_ten, decimal_pos);
    if (exp_ten > std::numeric_limits<double>::max_exponent10){
      out[row] = sign > 0 ? std::numeric_limits<double>::infinity()
                          : -std::numeric_limits<double>::infinity();
      return;
    }

    // make sure we don't produce a subnormal number. 
    // - a normal number is one where the leading digit of the floating point rep not zero. 
    //      eg:   0.0123  represented as  1.23e-2
    //
    // - a denormalized number is one where the leading digit of the floating point rep is zero.
    //      eg:   0.0123 represented as   0.123e-1
    //
    // - a subnormal number is a denormalized number where if you tried to normalize it, the exponent
    //   required would be smaller then the smallest representable exponent. 
    // 
    // https://en.wikipedia.org/wiki/Denormal_number
    // 

    if(warp_lane == 0){
      printf("row(%d), %lf, %d\n", row, digitsf, exp_ten);
    }

    double const exponent = exp10(static_cast<double>(std::abs(exp_ten)));
    double const result = exp_ten < 0 ? digitsf / exponent : digitsf * exponent;  
    
    out[row] = result;
  }    

  // compute_validity(tid, true);
}